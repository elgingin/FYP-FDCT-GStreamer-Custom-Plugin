#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

extern "C" void decompressFrame(hipfftComplex* input, float* output, int width, int height) {
    hipfftHandle plan;
    size_t size = width * height * sizeof(float);
    
    hipfftComplex* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, width * height * sizeof(hipfftComplex));
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, input, width * height * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    hipfftPlan2d(&plan, height, width, HIPFFT_C2R);
    hipfftExecC2R(plan, d_input, d_output);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);
}
